#include <stdio.h>

#include "hip/hip_runtime.h"

// 两个向量加法kernel，grid和block均为一维
__global__ void add(float* x, float* y, float* z, int n) {
    // 获取全局索引
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    // 步长
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride) {
        z[i] = x[i] + y[i];
    }
}

int main() {
    int N = 1 << 20;
    int nBytes = N * sizeof(float);

    // 申请托管内存
    float *x, *y, *z;
    hipMallocManaged((void**)&x, nBytes);
    hipMallocManaged((void**)&y, nBytes);
    hipMallocManaged((void**)&z, nBytes);

    // 初始化数据
    for (int i = 0; i < N; ++i) {
        x[i] = 10.0;
        y[i] = 20.0;
    }

    // 定义kernel的执行配置
    dim3 blockSize(256);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x);
    // 执行kernel
    add<<<gridSize, blockSize>>>(x, y, z, N);

    // 同步device 保证结果能正确访问
    hipDeviceSynchronize();
    // 检查执行结果
    float maxError = 0.0;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(z[i] - 30.0));
    printf("最大误差: %.2f\n", maxError);

    // 释放内存
    hipFree(x);
    hipFree(y);
    hipFree(z);

    return 0;
}